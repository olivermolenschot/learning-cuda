
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void vector_addition(float* A, float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i]; 
    }
}

int main() {
    // Host arrays
    float A[5] = {1, 2, 3, 4, 5};
    float B[5] = {1, 2, 3, 4, 5};
    float C[5]; // Result array
    int N = 5;

    // Device arrays
    float *d_a, *d_b, *d_c;

    // Allocate memory on the device
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_a, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with 1 block and N threads
    vector_addition<<<1, N>>>(d_a, d_b, d_c, N);

    // Copy result back from device to host
    hipMemcpy(C, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++) {
        cout << "C[" << i << "] = " << C[i] << endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
